// https://www.youtube.com/watch?v=2EbHSCvGFM0

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#define SIZE	1024

__global__ void invsqrt(double *a, double *b, int n)
{
	//int i = threadIdx.x;
	int i;
	for(i=threadIdx.x;i<n;i+=blockDim.x)
        {	
	    b[i] = 1.0/sqrt(a[i]);
	}
}

int main()
{
	double *a, *b;
        int i,num;
        num = 100000*SIZE;	
	hipMallocManaged(&a, num * sizeof(double)); // Memory for both CPU and GPU
	hipMallocManaged(&b, num * sizeof(double));
	
	for (i = 0; i < num; ++i)
	{
		a[i] = i;
		b[i] = 0.0;
	}
	
	invsqrt<<<1,SIZE>>>(a, b, num);
	hipDeviceSynchronize(); // Ensure CPU waits
        double tot;
	tot = 0.0;
	for (i = 1; i < num; ++i)
		tot+=b[i];
        printf("tot = %.17f\n",tot);

	hipFree(a);
	hipFree(b);

	return 0;
}

// https://www.youtube.com/watch?v=2EbHSCvGFM0

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define SIZE	1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;
	printf("blockDim.x = %d\n",blockDim.x);
	printf("gridDim.x = %d\n", gridDim.x);

	if (i<n)
	    c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;
	
	hipMallocManaged(&a, SIZE * sizeof(int)); // Memory for both CPU and GPU
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));
	
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	
	VectorAdd<<<1,SIZE>>>(a, b, c, SIZE);
	hipDeviceSynchronize(); // Ensure CPU waits

	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512 
__global__ void mykernel(void) 
{

}

__global__ void add(int *a, int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}


int main(void)
{

    int *a,*b,*c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    int i;
    hipMalloc( (void **)&d_a,size);
    hipMalloc( (void **)&d_b,size);
    hipMalloc( (void **)&d_c,size);
    
    a = (int *)malloc(size); 
    b = (int *)malloc(size); 
    c = (int *)malloc(size);

    for (i=0;i<N;i++) 
    {
        a[i] = rand();

        b[i] = rand();

    }
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1,N>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    for(i=0;i<100;i++)printf("%d\n",c[i]); 
    return 0;
}

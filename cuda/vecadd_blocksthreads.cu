#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512
__global__ void mykernel(void) 
{

}

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    c[index] = a[index] + b[index];
}


int main(void)
{

    int *a,*b,*c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    int i;
    hipMalloc( (void **)&d_a,size);
    hipMalloc( (void **)&d_b,size);
    hipMalloc( (void **)&d_c,size);
    
    a = (int *)malloc(size); 
    b = (int *)malloc(size); 
    c = (int *)malloc(size);

    for (i=0;i<N;i++) 
    {
        a[i] = rand();

        b[i] = rand();

    }
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    for(i=0;i<100;i++)printf("%d\n",c[i]); 
    return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) 
{

}

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}


int main(void)
{

    int a,b,c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    hipMalloc( (void **)&d_a,size);
    hipMalloc( (void **)&d_b,size);
    hipMalloc( (void **)&d_c,size);
    
    a = 2; b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a,d_b,d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    printf("%d\n",c);
    return 0;
}

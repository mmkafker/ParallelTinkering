// https://www.youtube.com/watch?v=2EbHSCvGFM0

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define SIZE	1024

__global__ void dotprod(int *a, int *b,int *c, int n)
{
	int i = threadIdx.x;
	__shared__ int d;
	d=0;
	if (i<n)
	{
	    c[i] = a[i]* b[i];
	    d += a[i]*b[i];
	}
	__syncthreads();
	printf("d = %d\n",d);

}

int main()
{
	int *a, *b, *c;
        int i;	
	hipMallocManaged(&a, SIZE * sizeof(int)); // Memory for both CPU and GPU
	hipMallocManaged(&b, SIZE * sizeof(int));
        	
	hipMallocManaged(&c, SIZE * sizeof(int));

	
	for (i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	
	dotprod<<<1,SIZE>>>(a, b, c,SIZE);
	hipDeviceSynchronize(); // Ensure CPU waits

	for (i=0;i<20;i++)printf("c[%d] = %d\n",i,c[i]);
	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}

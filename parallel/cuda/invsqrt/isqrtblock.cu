// https://www.youtube.com/watch?v=2EbHSCvGFM0

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#define SIZE	1024

#define DEBUG

__global__ void invsqrt(double *a, double *b, int n)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
        int stride = blockDim.x * gridDim.x;

	for(i= threadIdx.x + blockDim.x*blockIdx.x;i<n;i+=stride) b[i] = 1.0/sqrt(a[i]);
	
}

int main()
{
	double *a, *b, *d_a, *d_b;
        int i,num;
        num = 100000*SIZE;
        a = (double *)malloc(num * sizeof(double));

        b = (double *)malloc(num * sizeof(double));	
	hipMalloc((void **)&d_a, num * sizeof(double)); 
	hipMalloc((void **)&d_b, num * sizeof(double));
	
	for (i = 0; i < num; ++i)
	{
		a[i] = i;
		b[i] = 0.0;
	}

	hipMemcpy(d_a, a, num * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, num * sizeof(double), hipMemcpyHostToDevice);

	invsqrt<<<1000,SIZE>>>(d_a, d_b, num);
	hipDeviceSynchronize(); // Ensure CPU waits


	hipMemcpy(a, d_a, num * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, num * sizeof(double), hipMemcpyDeviceToHost);

//#ifndef DEBUG
        double tot;
	tot = 0.0;
	for (i = 1; i < num; ++i)
		tot+=b[i];
        printf("tot = %.17f\n",tot);

//#endif
	hipFree(d_a);
	hipFree(d_b);
        free(a); free(b);
	return 0;
}

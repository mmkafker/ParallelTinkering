#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000 
__global__ void stencil_1d(int *in, int *out)
{
    __shared__ int temp[BLOCK_SIZE + 2*RADIUS];
    int gindex = threadIdx.x + blockIdx.x*blockDim.x;
    int lindex = threadIdx.x + radis
}


__global__ void add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


int main(void)
{

    int *a,*b,*c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    int i;
    hipMalloc( (void **)&d_a,size);
    hipMalloc( (void **)&d_b,size);
    hipMalloc( (void **)&d_c,size);
    
    a = (int *)malloc(size); 
    b = (int *)malloc(size); 
    c = (int *)malloc(size);

    for (i=0;i<N;i++) 
    {
        a[i] = rand();

        b[i] = rand();

    }
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    for(i=0;i<100;i++)printf("%d\n",c[i]); 
    return 0;
}
